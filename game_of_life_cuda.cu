#include "hip/hip_runtime.h"
/**
 * Conway's Game of Life using Cuda
 * 
 * This version runs in parallel on a GPU using Cuda. Compile with:
 * 	   gcc -Wall -O3 -march=native -c util.c helpers.c
 *     nvcc -arch=sm_20 -O3 game_of_life_cuda.cu util.o -o game_of_life_cuda -lm
 * And run with:
 * 	   ./game_of_life_cuda num-of-iterations input-file output-file
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <inttypes.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/mman.h>

#include "util.h"

#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
      exit(1);                                                            \
   }                                                                      \
}

/**
 * Print the grid of organisms.
 */
void print_world(uint8_t* grid, size_t world_size) {
	for (size_t i = 0; i < world_size; i++) {
		for (size_t j = 0; j < world_size; j++) {
			printf("%s", grid[i*world_size+j] ? "█" : "-");
		}
		printf("\n");
	}
    printf("\n");
}

/**
 * Calculates the number of neighbors around an organism on the grid.
 */
__device__
int get_num_neighbors(uint8_t* grid, const size_t i, const size_t sz) {
    const size_t x = i % sz, y = i / sz;
    int neighbor_count = 0;

	// Check horizontally, vertically, and diagonally adjacent neighbors (as necessary)
    neighbor_count += x >= 1     && y >= 1     && grid[i-sz-1];
    neighbor_count +=               y >= 1     && grid[i-sz];
    neighbor_count += x < sz     && y >= 1     && grid[i-sz+1];
    neighbor_count += x >= 1                   && grid[i-1];
    neighbor_count += x < sz-1                 && grid[i+1]; 
    neighbor_count += x >= 1     && (y < sz-1) && grid[i+sz-1]; 
    neighbor_count +=                y < sz-1  && grid[i+sz]; 
    neighbor_count += (x < sz-1) && (y < sz-1) && grid[i+sz+1];

    return neighbor_count;
}

/**
 * Make the current position array the next position array.
 */
__device__
void swap(uint8_t** grid, uint8_t** grid_next) {
    uint8_t* temp = *grid;
    *grid = *grid_next;
    *grid_next = temp;
}

/**
 * The game of life simulation. Checks the number of neighbors each 
 * organism has each iteration and sets them to dead or alive.
 * Doesn't move onto next iteration until all threads are synchronized.
 */
__global__
void simulate(uint8_t* grid, uint8_t* grid_next, const size_t world_size, const size_t iterations) {
	const int i = threadIdx.x;
	const int grid_size = world_size * world_size;
	for (size_t step = 0; step < iterations; step++) {
		for (size_t k = i; k < grid_size; k += blockDim.x) {
			int n = get_num_neighbors(grid, k, world_size);
			grid_next[k] = grid[i] && (n > 1 && n <= 3) || !grid[i] && (n == 3);
		}
		swap(&grid, &grid_next);
		__syncthreads();
	}
}

/**
 * Benchmark function for determining how long memory copies take. 
 * This is used to determine how much time just memory operations take.
 */
 void cuda_memonly(const char* input_file) {
	size_t m, n;
	uint8_t* grid = grid_from_npy_path(input_file, &m, &n);
	if (!grid) { perror("grid_from_npy_path(grid)"); return; }

	// Allocate memory on the host
	size_t grid_size = m * n;
	const size_t grid_bytes = grid_size*sizeof(uint8_t);
	uint8_t* h_grid_copy = (uint8_t*) malloc(grid_bytes);
	uint8_t* h_grid_next = (uint8_t*) malloc(grid_bytes);
	memcpy(h_grid_copy, grid, grid_size);

	// Allocate memory on the device
	uint8_t *d_grid, *d_grid_next;
    CHECK(hipMalloc(&d_grid, grid_bytes));
    CHECK(hipMalloc(&d_grid_next, grid_bytes));

	// Copy memory from the host to the device and run the simulation
    CHECK(hipMemcpy(d_grid, h_grid_copy, grid_bytes, hipMemcpyHostToDevice));

	// Cleanup
	size_t addr = ((size_t)grid) & ~(sysconf(_SC_PAGE_SIZE)-1);
	munmap((void*)addr, grid_size*sizeof(uint8_t));
    free(h_grid_copy); free(h_grid_next);
    CHECK(hipFree(d_grid)); CHECK(hipFree(d_grid_next));
}

int main(int argc, char* const argv[]) {
	size_t iterations = 1000;
	const char * input_file = "examples/input.npy";
	const char * output_file = "output/out.npy";

	// Arguments can either be the number of iterations, the input/output file, or all 3
	if (argc > 4) { printf("Wrong number of arguments!\n"); return 1; }
	if (argc == 2) {
		iterations = atoi(argv[1]);
		if (iterations <= 0) { fprintf(stderr, "Must specify a positive number of iterations\n"); return 1; }
	} else if (argc == 3) {
		input_file = argv[1];
		output_file = argv[2];
	} else {
		iterations = atoi(argv[1]);
		input_file = argv[2];
		output_file = argv[3];
	}

	// Begin timing
	struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

	// Get the initial grid from the input file
	size_t m, n;
	uint8_t* grid = grid_from_npy_path(input_file, &m, &n);
	if (!grid) { perror("grid_from_npy_path(grid)"); return 1; }

	// Allocate memory on the host
	size_t grid_size = m * n;
	const size_t grid_bytes = grid_size*sizeof(uint8_t);
	uint8_t* h_grid_copy = (uint8_t*) malloc(grid_bytes);
	uint8_t* h_grid_next = (uint8_t*) malloc(grid_bytes);
	memcpy(h_grid_copy, grid, grid_size);

	// Allocate memory on the device
	uint8_t *d_grid, *d_grid_next;
    CHECK(hipMalloc(&d_grid, grid_bytes));
    CHECK(hipMalloc(&d_grid_next, grid_bytes));

	// Copy memory from the host to the device and run the simulation
    CHECK(hipMemcpy(d_grid, h_grid_copy, grid_bytes, hipMemcpyHostToDevice));
	int dimx = 1024, dimy = 1; 
    dim3 block(dimx, dimy);
    dim3 grid_cuda((m + dimx - 1) / dimx, (n + dimy - 1)/ dimy);
    simulate<<<grid_cuda, block>>>(d_grid, d_grid_next, n, iterations);
    CHECK(hipDeviceSynchronize());

	// Copy memory back from the device to the host and save to output file
    CHECK(hipMemcpy(h_grid_next, d_grid_next, grid_bytes, hipMemcpyDeviceToHost));
	
	// End timing
    clock_gettime(CLOCK_MONOTONIC, &end);
    double time = end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;
    printf("Time for complete simulation: %g secs\n", time);

	// Time memory allocations/copies only
	clock_gettime(CLOCK_MONOTONIC, &start);
	cuda_memonly(input_file);
	clock_gettime(CLOCK_MONOTONIC, &end);
 	double mem_time = end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;
    printf("Time for mem allocs/copies: %g secs\n", time);
	printf("Time running just on device: %g secs\n", time - mem_time);

	// Cleanup
	grid_to_npy_path(output_file, h_grid_next, 1, m, n);
	size_t addr = ((size_t)grid) & ~(sysconf(_SC_PAGE_SIZE)-1);
	munmap((void*)addr, grid_size*sizeof(uint8_t));
    free(h_grid_copy); free(h_grid_next);
    CHECK(hipFree(d_grid)); CHECK(hipFree(d_grid_next));
    return 0;
}
